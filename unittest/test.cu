#include <gpu_bsw/kernel.hpp>
#include <gpu_bsw/utils.hpp>

#include <thrust/device_vector.h>

#include <iostream>
#include <random>
#include <vector>

using namespace gpu_bsw;

std::mt19937 gen;
std::uniform_int_distribution<short> dna_dist(0,3);
std::uniform_int_distribution<short> len_dist(1,32);

std::vector<char> randomDNA(const int len){
  constexpr std::array<char, 4> bases = {'A','C','G','T'};
  std::vector<char> temp;
  temp.reserve(len);
  for(int i=0;i<len;i++){
    temp.push_back(bases.at(dna_dist(gen)));
  }
  return temp;
}



template<class T>
std::ostream& operator<<(std::ostream &out, thrust::device_vector<T> &vec){
  for(size_t i=0;i<vec.size();i++)
    out<<vec[i]<<" ";
  return out;
}



int main(){
  const int alen = 40;
  const int blen = 100;

  thrust::device_vector<char> seqA_array;
  seqA_array = randomDNA( alen);
  thrust::device_vector<char> seqB_array;
  seqB_array = randomDNA(blen);
  thrust::device_vector<unsigned> prefix_lengthA(1);
  thrust::device_vector<unsigned> prefix_lengthB(1);

  prefix_lengthA[0] = alen;
  prefix_lengthB[0] = blen;

  thrust::device_vector<short> seqA_align_begin(1);
  thrust::device_vector<short> seqA_align_end(1);
  thrust::device_vector<short> seqB_align_begin(1);
  thrust::device_vector<short> seqB_align_end(1);

  thrust::device_vector<short> top_scores(1);

  thrust::device_vector<short> scoring_matrix(2);
  thrust::device_vector<short> encoding_matrix(1);

  scoring_matrix[0] = 1;
  scoring_matrix[1] = -4;

  encoding_matrix[0] = 0;

  const short startGap = 6;
  const short extendGap = 1;

  // unsigned minSize = (maxReadSize < maxContigSize) ? maxReadSize : maxContigSize;
  const unsigned minSize = alen;
  const unsigned totShmem = 3 * (minSize + 1) * sizeof(short);
  const unsigned alignmentPad = 4 + (4 - totShmem % 4);
  const size_t   ShmemBytes = totShmem + alignmentPad;
  // if(ShmemBytes > 48000 && DT==DataType::DNA)
      // hipFuncSetAttribute(reinterpret_cast<const void*>(gpu_bsw::sequence_process<DataType::DNA),Direction::FORWARD>, hipFuncAttributeMaxDynamicSharedMemorySize, ShmemBytes);

  cudaErrchk(hipDeviceSynchronize());

  sequence_process<DataType::DNA, Direction::FORWARD><<<1, minSize, ShmemBytes>>>(
    thrust::raw_pointer_cast(seqA_array.data()),
    thrust::raw_pointer_cast(seqB_array.data()),
    thrust::raw_pointer_cast(prefix_lengthA.data()),
    thrust::raw_pointer_cast(prefix_lengthB.data()),
    thrust::raw_pointer_cast(seqA_align_begin.data()),
    thrust::raw_pointer_cast(seqA_align_end.data()),
    thrust::raw_pointer_cast(seqB_align_begin.data()),
    thrust::raw_pointer_cast(seqB_align_end.data()),
    thrust::raw_pointer_cast(top_scores.data()),
    startGap,
    extendGap,
    thrust::raw_pointer_cast(scoring_matrix.data()),
    thrust::raw_pointer_cast(encoding_matrix.data())
  );
  cudaErrchk(hipGetLastError());

  cudaErrchk(hipDeviceSynchronize());

  std::cout<<"seqA             = "<<seqA_array<<"\n";
  std::cout<<"seqB             = "<<seqB_array<<"\n";
  std::cout<<"seqA_align_begin = "<<seqA_align_begin<<"\n";
  std::cout<<"seqA_align_end   = "<<seqA_align_end<<"\n";
  std::cout<<"seqB_align_begin = "<<seqB_align_begin<<"\n";
  std::cout<<"seqB_align_end   = "<<seqB_align_end<<"\n";
  std::cout<<"top_scores       = "<<top_scores<<"\n";
}